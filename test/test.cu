/*
 * test.cu
 *
 * This library is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 3.0 of the License, or (at your option) any later version.
 *
 * This library is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU 
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with this library; If not, see <http://www.gnu.org/licenses/>.
 */
#include <pga.h>
#include <limits.h>
#include <float.h>
#include <stdio.h>

#define GENOME_LENGTH 100

__device__ float objf(gene *g, unsigned length) {
	float s = 0;
	for (int i = 0; i < length; ++i) {
		s += g[i];
	}
	return s;
}

__device__ obj_f ofunction = objf;

int main() {
	pga_t *p = pga_init();

	population_t *pop = pga_create_population(p, 40000, GENOME_LENGTH, RANDOM_POPULATION);

	void *func;
	hipMemcpyFromSymbol( &func , HIP_SYMBOL(ofunction ), sizeof(obj_f));
	pga_set_objective_function(p, (obj_f)func);
	
	pga_run(p, 100);
	
	pga_get_best(p, pop);
	
	pga_deinit(p);
	return 0;
}
